#include "hip/hip_runtime.h"
#define NPY_NO_DEPRECATED_API NPY_1_7_API_VERSION
#include "../gpuassert.cu"
#include "numpy/ndarraytypes.h"
#include <Python.h>
#include <numpy/arrayobject.h>
#include <time.h>

// change to npy_float64 for more precision
typedef npy_float32 cufloat;
const int CUFLOAT = NPY_FLOAT32;

// Calculates the dot product sum for a single complex coefficient.
// Run with a dim3.
__global__ void matmul_elem(const npy_intp N, const cufloat *a,
                            const cufloat *b, cufloat *dest) {
  const int row = blockIdx.x * blockDim.x + threadIdx.x;
  const int column = blockIdx.y * blockDim.y + threadIdx.y;

  if (column < N && row < N) {
    double dot_product = 0.f;

    for (int i = 0; i < N; i++) {
      dot_product += a[row * N + i] * b[i * N + column];
    }

    dest[row * N + column] = dot_product;
  }
}

// Calculates the dot product sum for a single complex coefficient.
// Run with a dim1. Assumes b is diagonal.
// PERF : Could probably benefit from better caching by idexing by column
// instead of row, so the destination matrix would be filled row by row.
__global__ void matmul_diag_elem(const npy_intp N, const cufloat *a,
                                 const cufloat *b, cufloat *dest) {
  const int row = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < N) {
    for (int column = 0; column < N; column++) {
      const cufloat a_i = a[row * N + column];
      const cufloat b_i = b[column * N + column];

      double dot_product = a_i * b_i;

      dest[row * N + column] = dot_product;
    }
  }
}

/**
 * Multiply A * D * Ainv square matrices of dimensions N * N
 *
 * D is a diagonal matrix.
 * Ainv is the inverse matrix of A.
 *
 * NOTE : not an optimal implementation
 */
static void matmul_ADAinv_gpu(const npy_intp N, const cufloat *a,
                              const cufloat *d, const cufloat *ainv,
                              cufloat *dest) {
  printf("[matmul] Allocating and copying to device...\n");
  clock_t start = clock();

  // data on device
  cufloat *a_d, *d_d, *ainv_d, *sum_d;
  hipMalloc((void **)&a_d, N * N * sizeof(cufloat));
  hipMalloc((void **)&d_d,
             N * N * sizeof(cufloat)); // could just be a vec
  hipMalloc((void **)&ainv_d, N * N * sizeof(cufloat));
  hipMalloc((void **)&sum_d, N * N * sizeof(cufloat));

  // copy data from host to device
  hipMemcpy(a_d, a, N * N * sizeof(cufloat), hipMemcpyHostToDevice);
  hipMemcpy(d_d, d, N * N * sizeof(cufloat), hipMemcpyHostToDevice);
  hipMemcpy(ainv_d, ainv, N * N * sizeof(cufloat), hipMemcpyHostToDevice);
  gpuErrchk(hipPeekAtLastError());
  gpuErrchk(hipDeviceSynchronize());

  clock_t end_alloc = (clock() - start) * 1000 / CLOCKS_PER_SEC;
  printf("[matmul] Copied data to device. Calculating...\n");

  // calculate sum = A * D
  const int BLOCK_SIZE_DIAG = 1024; // should be <= 1024 I think
  matmul_diag_elem<<<ceil(N / (float)BLOCK_SIZE_DIAG), BLOCK_SIZE_DIAG>>>(
      N, a_d, d_d, sum_d);
  gpuErrchk(hipPeekAtLastError());
  gpuErrchk(hipDeviceSynchronize());

  clock_t end_diag = (clock() - start) * 1000 / CLOCKS_PER_SEC;

  // calculate A = sum * Ainv
  const int BLOCK_SIZE = 32; // because 32**2 = 1024 threads
  dim3 dimGrid(ceil(N / (float)BLOCK_SIZE), ceil(N / (float)BLOCK_SIZE), 1);
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
  matmul_elem<<<dimGrid, dimBlock>>>(N, sum_d, ainv_d, a_d);
  gpuErrchk(hipPeekAtLastError());
  gpuErrchk(hipDeviceSynchronize());

  clock_t end_mul = (clock() - start) * 1000 / CLOCKS_PER_SEC;

  printf("[matmul] Done calculating, retrieving data and freeing...\n");

  // copy data from devices to host
  hipMemcpy(dest, a_d, N * N * sizeof(cufloat), hipMemcpyDeviceToHost);

  hipFree(a_d);
  hipFree(d_d);
  hipFree(ainv_d);
  hipFree(sum_d);

  clock_t end_free = (clock() - start) * 1000 / CLOCKS_PER_SEC;
  printf("[matmul] Done. alloc=%ldms, diag=%ldms, mul=%ldms, free=%ldms\n",
         end_alloc, end_diag, end_mul, end_free);
}

// Function to perform an operation on a complex NumPy array
static PyObject *complex_operation(PyObject *self, PyObject *args) {
  PyObject *input_array;

  // Parse the input arguments
  if (!PyArg_ParseTuple(args, "O", &input_array)) {
    return NULL;
  }

  // Ensure the input is a NumPy array
  if (!PyArray_Check(input_array)) {
    PyErr_SetString(PyExc_TypeError, "Input must be a NumPy array");
    return NULL;
  }

  // Get the array and ensure it is of complex type
  // increases refcount to array.
  PyArrayObject *array = (PyArrayObject *)PyArray_FROM_OTF(
      input_array, CUFLOAT, NPY_ARRAY_WRITEABLE | NPY_ARRAY_C_CONTIGUOUS);
  if (array == NULL) {
    PyErr_SetString(PyExc_TypeError, "Input array must contain float32 values");
    return NULL;
  }

  // Get the dimensions of the array
  int dimension_count = PyArray_NDIM(array);
  npy_intp *dimensions = PyArray_DIMS(array);
  npy_intp size = PyArray_SIZE(array);

  // Ensure the matrix is square
  if (dimension_count != 2 || dimensions[0] != dimensions[1]) {
    PyErr_SetString(PyExc_TypeError, "Input must be a square matrix");
    return NULL;
  }

  // Get a pointer to the data
  const cufloat *data = (cufloat *)PyArray_DATA(array);

  // Create result matrix
  PyObject *result_matrix_object =
      PyArray_NewLikeArray(array, NPY_CORDER, NULL, 1);
  PyArrayObject *result_matrix_array = (PyArrayObject *)result_matrix_object;
  // Get a pointer to the data
  cufloat *result_matrix = (cufloat *)PyArray_DATA(result_matrix_array);

  matmul_ADAinv_gpu(dimensions[0], data, data, data, result_matrix);

  // printf("Refcount to input_array before free: %d\n",
  // Py_REFCNT(input_array)); printf("Refcount to array before free: %d\n",
  // Py_REFCNT(array));

  // Decrease the reference count of the input array
  Py_DECREF(array);
  return result_matrix_object;
}

// Method definitions
static PyMethodDef ComplexMethods[] = {
    {"complex_operation", complex_operation, METH_VARARGS,
     "Perform an operation on a complex NumPy array"},
    {NULL, NULL, 0, NULL}};

// Module definition
static struct PyModuleDef complexmodule = {
    PyModuleDef_HEAD_INIT, "complexmodule",
    "Module for performing operations on complex NumPy arrays", -1,
    ComplexMethods};

// Module initialization
PyMODINIT_FUNC PyInit_complexmodule(void) {
  PyObject *m;
  import_array(); // Initialize NumPy API
  m = PyModule_Create(&complexmodule);
  return m;
}
